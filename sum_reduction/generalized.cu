// This program performs sum reduction with an optimization
// removing warp bank conflicts
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <vector>

#define SIZE 256

__global__ void sum_reduction(int *v, int *v_r, int n) {
  // Calculate thread ID
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Boundary check
  if (tid < n) {
    // Allocate shared memory
    __shared__ int partial_sum[SIZE];

    // Calculate the number of elements this block reduces
    // Only the last block may have stragglers
    int reduce_elements;
    if (blockIdx.x == gridDim.x - 1) {
      reduce_elements = n - blockIdx.x * SIZE;
    } else {
      reduce_elements = SIZE;
    }

    // Find the next power of two
    // __clz finds the leading number of zeros in an int
    int next_power = 1 << (32 - __clz(reduce_elements) + 1);
    int init = next_power > SIZE ? SIZE : next_power;

    // Load elements into shared memory
    partial_sum[threadIdx.x] = v[tid];
    __syncthreads();

    // Start with a padded number of reduce_elements
    for (int s = init; s > 0; s >>= 1) {
      // Only threads < stride compute partial sums
      // Only threads accessing elements < reduce_elements need to be active
      // This handles the case where reduce_elements is an odd number
      if (threadIdx.x < s && threadIdx.x + s < reduce_elements) {
        partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
      }
      __syncthreads();
    }

    // Let the thread 0 for this block write it's result to main memory
    // Result is inexed by this block
    if (threadIdx.x == 0) {
      v_r[blockIdx.x] = partial_sum[0];
    }
  }
}

int main() {
  // Vector size
  int n = 1 << 20;
  size_t bytes = n * sizeof(int);

  // Host-side input vector
  std::vector<int> h_v(n);
  std::generate(begin(h_v), end(h_v), []() { return 1; });

  // Single result element
  int h_v_r = 0;

  // Allocate device memory
  int *d_v, *d_v_r;
  hipMalloc(&d_v, bytes);
  hipMalloc(&d_v_r, bytes);

  // Copy to device
  hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);

  // TB Size
  int TB_SIZE = SIZE;

  // Grid Size
  int GRID_SIZE = (n + TB_SIZE - 1) / TB_SIZE;

  // Number of elements reduced in the next iteration
  int num_elements = n;

  // Launch kernels until we've performed the complete reduction
  while (1) {
    // Call kernel
    sum_reduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r, num_elements);

    // No more reductions left!
    if (GRID_SIZE == 1) break;

    // Swap the pointers each iteration
    // Output from last iteration is the input to the next
    std::swap(d_v, d_v_r);

    // Calculate the number of elements next iteration
    // Number of input elements next iteration is the number of output
    // elements from last iteration
    num_elements = GRID_SIZE;

    // Calculate padded grid size
    GRID_SIZE = (GRID_SIZE + TB_SIZE - 1) / TB_SIZE;
  }

  // Copy the result back to the host
  hipMemcpy(&h_v_r, d_v_r, sizeof(int), hipMemcpyDeviceToHost);

  // Host sum
  int res = std::accumulate(begin(h_v), end(h_v), 0);

  // Check the result
  assert(h_v_r == res);

  std::cout << "COMPLETED SUCCESSFULLY!\n";

  return 0;
}
