// This program implements a simple, but flexible sum reduction
// implementation in CUDA
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>

// Each TB needs some shared memory
// Allocate for 256 ints
#define SHMEM_SIZE 256 * 4

using namespace std;

// Sum reduction kernel taken from previous CUDA video
// Slightly modified to handle inputs of not powers of 256
__global__ void sum_reduction(int *v_in, int *v_out, int N){
	// Allocate shared memory statically
	__shared__ int partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
   
    // Mask off inactive threads in a TB
    if(tid < N){
	    // Load elements into shared memory
	    partial_sum[threadIdx.x] = v_in[tid];
	    __syncthreads();

        // How many elements we sum in this TB depends on how many remaining
        // elements there are. If 256, use blockDim, else, use N
        int max = (N < blockDim.x) ? N : blockDim.x;

	    // Iterate of log base 2 the block dimension
	    for (int s = 1; s < max; s *= 2) {
		    // Reduce the threads performing work by half previous the previous
		    // iteration each cycle
		    if (threadIdx.x % (2 * s) == 0) {
			    partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		    }
		    __syncthreads();
	    }

	    // Let the thread 0 for this block write it's result to main memory
	    // Result is inexed by this block
	    if (threadIdx.x == 0) {
		    v_out[blockIdx.x] = partial_sum[0];
	    }
    }
}

int main(){
    // Number of elements
    int N = 1 << 20;
    int CPU_SUM = N;
    size_t bytes = N * sizeof(int);

    // Host arrays
    int *h_v_in = new int[N];
    int *h_v_out = new int[1];

    // Device arrays
    int *d_v_in, *d_v_out;
    hipMalloc(&d_v_in, bytes);
    hipMalloc(&d_v_out, bytes);

    // Init input array
    for(int i = 0; i < N; i++){
        h_v_in[i] = 1;
    }

    // Copy the array over
    hipMemcpy(d_v_in, h_v_in, bytes, hipMemcpyHostToDevice);

    // TB size;
    int THREADS = 256;

    // Track the number of iterations it takes
    int iter = 0;
    
    // Grid size may change each loop iteration
    int GRID;

    // Simple loop to keep launching kernels until we're done
    // N == 1 means we only have 1 elements left, aka, we're done
    // If it's not a power of 256, N will converge to 0 instead
    while(N > 1){
        // Calculate the grid size
        GRID = (N + THREADS - 1) / THREADS;
        
        // Alternate kernel inputs
        // This is so we don't need to constantly re-allocate new
        // output arrays
        if(iter % 2){
            sum_reduction<<<GRID, THREADS>>>(d_v_out, d_v_in, N);
        }else{
            sum_reduction<<<GRID, THREADS>>>(d_v_in, d_v_out, N);
        }

        // LOG_256(N) iterations of the loop
        iter++;
        N /= 256;
    }

    // Which array we copy back from depends on the final iter #
    // Only a single sum needs to be copied out
    if(iter % 2){
        hipMemcpy(h_v_out, d_v_out, sizeof(int), hipMemcpyDeviceToHost);
    }else{
        hipMemcpy(h_v_out, d_v_in, sizeof(int), hipMemcpyDeviceToHost);
    }

    // Print the result
    cout << "Number of iterations = " << iter << endl;
    cout << "Reduced Sum = " << h_v_out[0] << endl;
    cout << "CPU sum = " << CPU_SUM << endl;

    return 0;
}
